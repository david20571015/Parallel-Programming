#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "common/CycleTimer.h"

__device__ int mandel(float c_re, float c_im, int count) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f) break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, int* img, float stepX,
                             float stepY, int resX, int resY,
                             int maxIterations) {
  int thisX = blockIdx.x * blockDim.x + threadIdx.x;
  int thisY = blockIdx.y * blockDim.y + threadIdx.y;

  float x = lowerX + stepX * thisX;
  float y = lowerY + stepY * thisY;

  int* ptr = (int*)(img + thisY * resX + thisX);
  *ptr = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;

  static constexpr int BLOCK_SIZE = 16;
  dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_size(resX / BLOCK_SIZE, resY / BLOCK_SIZE);

  // Allocate memory on the host
  int* h_img = (int*)malloc(sizeof(int) * resX * resY);

  // Allocate memory on the device
  int* d_img;
  hipMalloc((void**)&d_img, sizeof(int) * resX * resY);

  // Launch the kernel
  mandelKernel<<<grid_size, block_size>>>(lowerX, lowerY, d_img, stepX, stepY,
                                          resX, resY, maxIterations);

  // Copy the result from the device to the host
  hipMemcpy(h_img, d_img, sizeof(int) * resX * resY, hipMemcpyDeviceToHost);
  hipFree(d_img);

  memcpy(img, h_img, sizeof(int) * resX * resY);
  free(h_img);
}
